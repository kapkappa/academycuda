// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>


//#define STREAMS_NUM 8
#define BLOCKSIZE 1024

int STREAMS_NUM;

__global__ void vectorAddGPU(float *a, float *b, float *c, int N, int offset) {

    int idx = blockIdx.x*blockDim.x + threadIdx.x + offset;
    if (idx < N)
        c[idx] = a[idx] + b[idx];
}

void sample_vec_add(int size = 1048576) {
    int n = size;
    int nBytes = n*sizeof(int);

    float *a, *b;  // host data
    float *c;  // results

    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);

    float *a_d,*b_d,*c_d;

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("Allocating device memory on host..\n");

    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));

    printf("Copying to device..\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);

    printf("Doing GPU Vector add\n");

    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n, 0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);

    hipDeviceSynchronize();

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void streams_vec_add(int size = 1048576) {
    int n = size;
    int nBytes = n*sizeof(int);

    float *a, *b;  // host data
    float *c;  // results

    hipHostAlloc( (void**) &a, n * sizeof(float) ,hipHostMallocDefault );
    hipHostAlloc( (void**) &b, n * sizeof(float) ,hipHostMallocDefault );
    hipHostAlloc( (void**) &c, n * sizeof(float) ,hipHostMallocDefault );

    float *a_d,*b_d,*c_d;

    for(int i=0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("Allocating device memory on host..\n");

    hipMalloc((void **)&a_d, n*sizeof(float));
    hipMalloc((void **)&b_d, n*sizeof(float));
    hipMalloc((void **)&c_d, n*sizeof(float));

    printf("Copying to device..\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    printf("Doing GPU Vector add\n");

    const int StreamSize = n / STREAMS_NUM;
    hipStream_t Stream[STREAMS_NUM];

    for (int i = 0; i < STREAMS_NUM; i++)
        hipStreamCreate(&Stream[i]);

    dim3 block(BLOCKSIZE);
    dim3 grid((StreamSize - 1)/BLOCKSIZE + 1);

    for (int i = 0; i < STREAMS_NUM; i++) {

        int Offset = i * StreamSize;

        hipMemcpyAsync(&a_d[Offset], &a[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[i]);
        hipMemcpyAsync(&b_d[Offset], &b[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[i]);
        hipMemcpyAsync(&c_d[Offset], &c[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[i]);

        vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, StreamSize, Offset);

        hipMemcpyAsync(&a[Offset], &a_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[i]);
        hipMemcpyAsync(&b[Offset], &b_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[i]);
        hipMemcpyAsync(&c[Offset], &c_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[i]);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);

    hipDeviceSynchronize();

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}


int main(int argc, char **argv) {
    assert(argc==4);
    STREAMS_NUM = atoi(argv[3]);
    if (atoi(argv[2]) == 0)
    	sample_vec_add(atoi(argv[1]));
    else
        streams_vec_add(atoi(argv[1]));
}
