#include <iostream>
#include <assert.h>
#include <cstdlib>
#include <hip/hip_runtime.h>


const int SIZE = 32;

__global__ void transpose(int *V, int n) {
//    __shared__ int array[SIZE];

    int Idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (Idx <=  n/2) {
        int tmp = V[Idx];
        V[Idx] = V[n-Idx-1];
        V[n-Idx-1] = tmp;
    }
}

int main(int argc, char **argv) {
    assert(argc==2);
    int n = atoi(argv[1]);

    size_t size = n * sizeof(int);
    int *V = (int*)malloc(size);

    for (int i = 0; i < n; i++) {
        V[i] = i;
    }

    int block = 1024;
    int grid = (n / 2 - 1) / block + 1;

    int *V_t;
    hipMalloc(&V_t, size);
    hipMemcpy(V_t, V, size, hipMemcpyHostToDevice);

    transpose<<<grid, block>>>(V_t, n);

    hipDeviceSynchronize();

    hipMemcpy(V, V_t, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        std::cout << V[i] << " ";
    }
    std::cout << std::endl;

    free(V);
    hipFree(V_t);

    return 0;
}
