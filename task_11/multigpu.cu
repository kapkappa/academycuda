#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 256

inline
hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void vectorAddGPU(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_sample (int size = 1048576) {
    int n = size;
    int nBytes = n * sizeof(float);

    float *a, *b, *c;

    hipEvent_t uniStart, uniStop;
    hipEventCreate(&uniStart);
    hipEventCreate(&uniStop);

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);
    hipMallocManaged(&c, nBytes);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEventRecord(uniStart);

    vectorAddGPU<<<grid, block>>>(a, b, c, n);

    hipEventRecord(uniStop);
    hipEventSynchronize(uniStop);
    float ms = 0;
    hipEventElapsedTime(&ms, uniStart, uniStop);
    printf("UNI: Memalloc(unified memory) + Kernel time is: %f\n", ms);

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(c);
}

void pinned_sample (int size = 1048576) {
    int n = size;
    size_t nBytes = n * sizeof(float);
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
//    float errNorm, refNorm, ref, diff;

    hipEvent_t pinStart, pinStop;
    hipEventCreate(&pinStart);
    hipEventCreate(&pinStop);

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);
    hipMalloc(&d_a, nBytes);
    hipMalloc(&d_b, nBytes);
    hipMalloc(&d_c, nBytes);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEventRecord(pinStart);

    hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, nBytes, hipMemcpyHostToDevice);
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, nBytes, hipMemcpyDeviceToHost);

    hipEventRecord(pinStop);
    hipDeviceSynchronize();
    float ms = 0;
    hipEventElapsedTime(&ms, pinStart, pinStop);
    printf("PIN: Memcpy + Kernel time is: %f\n", ms);

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);
}

void usual_sample (int size = 1048576) {
    int n = size;
    int nBytes = n*sizeof(float);

    float *a, *b;  // host data
    float *c;  // results

    a = (float*)malloc(nBytes);
    b = (float*)malloc(nBytes);
    c = (float*)malloc(nBytes);

    float *a_d,*b_d,*c_d;

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("Allocating device memory on host..\n");

    hipEvent_t start, stop, malloc_start, malloc_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
//    hipEventCreate(&malloc_start);
//    hipEventCreate(&malloc_stop);

    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));

    hipEventRecord(start);

    hipMemcpy(a_d, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(float), hipMemcpyHostToDevice);

    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    hipMemcpy(c, c_d, n*sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("USUAL Memcpy + Kernel: %f ms\n", milliseconds);

    hipDeviceSynchronize();

    free(a);
    free(b);
    free(c);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void multigpu (int size) {
    printf("DEBUG0\n");
    int n = size;
    int nBytes = n * sizeof(float);

    dim3 block(BLOCKSIZE);

    printf("DEBUG1\n");

    float *a, *b, *c;
    hipHostAlloc((void**)&a, nBytes, hipHostMallocPortable);
    hipHostAlloc((void**)&b, nBytes, hipHostMallocPortable);
    hipHostAlloc((void**)&c, nBytes, hipHostMallocPortable);

    printf("DEBUG2\n");

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("DEBUG3\n");

    int deviceCnt;
    hipGetDeviceCount(&deviceCnt);

    int bytes_per_device = nBytes / deviceCnt + 1;

    float *a_d[2], *b_d[2], *c_d[2];

    dim3 grid(n/(deviceCnt*BLOCKSIZE) + 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//    hipEventRecord(start);

    printf("\nParallel starts here\n\n");

    for (int i = 0; i < deviceCnt; i++) {
        hipSetDevice(i);
        hipMalloc(&a_d[i], bytes_per_device);
        hipMalloc(&b_d[i], bytes_per_device);
        hipMalloc(&c_d[i], bytes_per_device);
    }

    for (int i = 0; i < deviceCnt; i++) {
        hipSetDevice(i);
        hipMemcpy(a_d[i], a, bytes_per_device, hipMemcpyHostToDevice);
        hipMemcpy(b_d[i], b, bytes_per_device, hipMemcpyHostToDevice);

        vectorAddGPU<<<block, grid>>>(a_d[i], b_d[i], c_d[i], n);

        hipMemcpy(c, c_d[i], bytes_per_device, hipMemcpyDeviceToHost);
    }

    for (int dev = 0; dev < deviceCnt; dev++) {
        hipSetDevice(dev);
        hipDeviceSynchronize();
    }

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    for (int i = 0; i < deviceCnt; i++) {
        hipFree(a_d[i]);
        hipFree(b_d[i]);
        hipFree(c_d[i]);
    }
}

int main(int argc, char **argv) {
    assert(argc==3);
    switch (atoi(argv[2])) {
    case 0:
        usual_sample(atoi(argv[1]));
        break;
    case 1:
        pinned_sample(atoi(argv[1]));
        break;
    case 2:
        unified_sample(atoi(argv[1]));
        break;
    case 3:
        multigpu(atoi(argv[1]));
        break;
    default:
        break;
    }
    return 0;
}
