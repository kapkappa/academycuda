// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>


#include <omp.h>

#define BLOCKSIZE 1024

const float EPS = 1.e-06;

inline
hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void vectorAddGPU(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_sample (int size = 1048576) {
    int n = size;
    int nBytes = n * sizeof(float);

    float *a, *b, *c;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);
    hipMallocManaged(&c, nBytes);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEventRecord(start);

    vectorAddGPU<<<grid, block>>>(a, b, c, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("UNI: Memalloc(unified memory) + Kernel time is: %f\n", ms);

    hipDeviceSynchronize();

    printf("NO CHECK!\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

void pinned_sample (int size = 1048576) {
    int n = size;
    size_t nBytes = n * sizeof(float);
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
//    float errNorm, refNorm, ref, diff;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    hipHostMalloc(&a, nBytes, hipHostMallocDefault);
    hipHostMalloc(&b, nBytes, hipHostMallocDefault);
    hipHostMalloc(&c, nBytes, hipHostMallocDefault);
    hipMalloc(&d_a, nBytes);
    hipMalloc(&d_b, nBytes);
    hipMalloc(&d_c, nBytes);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEventRecord(start);

    hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, nBytes, hipMemcpyHostToDevice);
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, nBytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("PIN: Memcpy + Kernel time is: %f\n", ms);

    hipDeviceSynchronize();

    //CHECK
    for (int i = 0; i < n; i++) {
        if (abs(c[i] - a[i] - b[i]) > EPS) {
            printf("CHECK FAILED!!!\n");
            printf("Differ: %f\n", c[i]-a[i]-b[i]);
            break;
        }
    }
    printf("CHECK PASSED!\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);
}

void usual_sample (int size = 1048576) {
    int n = size;
    int nBytes = n*sizeof(float);

    float *a, *b;  // host data
    float *c;  // results

    a = (float*)malloc(nBytes);
    b = (float*)malloc(nBytes);
    c = (float*)malloc(nBytes);

    float *a_d,*b_d,*c_d;

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("Allocating device memory on host..\n");

    hipEvent_t start, stop, malloc_start, malloc_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
//    cudaEventCreate(&malloc_start);
//    cudaEventCreate(&malloc_stop);

    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));

    hipEventRecord(start);

    hipMemcpy(a_d, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(float), hipMemcpyHostToDevice);

    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    hipMemcpy(c, c_d, n*sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("USUAL Memcpy + Kernel: %f ms\n", milliseconds);

    hipDeviceSynchronize();

    //CHECK
    for (int i = 0; i < n; i++) {
        if (abs(c[i] - a[i] - b[i]) > EPS) {
            printf("CHECK FAILED!!!\n");
            printf("Differ: %f\n", c[i]-a[i]-b[i]);
            break;
        }
    }
    printf("CHECK PASSED!\n");

    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void multigpu (int size) {
    int n = size;
    int nBytes = n * sizeof(float);

    dim3 block(BLOCKSIZE);

    float *a, *b, *c;
    hipHostAlloc((void**)&a, nBytes, hipHostMallocDefault);
    hipHostAlloc((void**)&b, nBytes, hipHostMallocDefault);
    hipHostAlloc((void**)&c, nBytes, hipHostMallocDefault);

    for (int i = 0; i < n; i++) {
//        a[i] = rand() / (float)RAND_MAX;
//        b[i] = rand() / (float)RAND_MAX;
        a[i] = i;
        b[i] = 1;
        c[i] = 0;
    }

    int deviceCnt;
    hipGetDeviceCount(&deviceCnt);
      deviceCnt = 2;
    printf("Devices: %d", deviceCnt);

    int bytes_per_device = nBytes / deviceCnt + 1;

//    float (*a_d)[deviceCnt], (*b_d)[deviceCnt], (*c_d)[deviceCnt];
    float **a_d, **b_d, **c_d;
    a_d = (float**)malloc(sizeof(float*) * deviceCnt);
    b_d = (float**)malloc(sizeof(float*) * deviceCnt);
    c_d = (float**)malloc(sizeof(float*) * deviceCnt);


    int n_per_device = (n-1)/deviceCnt + 1;
    dim3 grid((n_per_device-1)/BLOCKSIZE + 1);

    hipEvent_t start[deviceCnt], stop[deviceCnt];
    for (int i = 0; i < deviceCnt; i++) {
        hipEventCreate(&start[i]);
        hipEventCreate(&stop[i]);
    }

    printf("\nParallel starts here\n\n");

    for (int i = 0; i < deviceCnt; i++) {
        hipSetDevice(i);
        int check;
        hipGetDevice(&check);
        assert(i == check);
        hipMalloc((void**)&(a_d[i]), bytes_per_device);
        hipMalloc((void**)&(b_d[i]), bytes_per_device);
        hipMalloc((void**)&(c_d[i]), bytes_per_device);
    }

    for (int i = 0; i < deviceCnt; i++) {
        hipSetDevice(i);
//        int check;
//        cudaGetDevice(&check);
//        assert(i == check);
        printf("i am on device %d\nposition is %d\n", i, n_per_device*i);
        hipEventRecord(start[i]);
        hipMemcpyAsync(a_d[i], a + n_per_device * i, bytes_per_device, hipMemcpyHostToDevice);
        hipMemcpyAsync(b_d[i], b + n_per_device * i, bytes_per_device, hipMemcpyHostToDevice);

        vectorAddGPU<<<grid, block>>>(a_d[i], b_d[i], c_d[i], n);

        hipMemcpyAsync(c + n_per_device * i, c_d[i], bytes_per_device, hipMemcpyDeviceToHost);
        hipEventRecord(stop[i]);
    }

    for (int dev = 0; dev < deviceCnt; dev++) {
        hipSetDevice(dev);
        hipDeviceSynchronize();
    }

    for (int i = 0; i < deviceCnt; i++) {
        float ms = 0;
        hipEventElapsedTime(&ms, start[i], stop[i]);
        printf("Elapsed time on device %d is %f\n", i, ms);
    }

    for (int i = 0; i < deviceCnt; i++) {
        hipEventDestroy(start[i]);
        hipEventDestroy(stop[i]);
    }
/*
    for (int i = 0; i < n; i++)
        printf("%f ", c[i]);
    printf("\n\n");
*/
    //CHECK
    for (int i = 0; i < n; i++) {
        if (abs(c[i] - a[i] - b[i]) > EPS) {
            printf("CHECK FAILED!!!\n");
            printf("Differ: %f\n", c[i]-a[i]-b[i]);
            break;
        }
    }
    printf("CHECK PASSED!\n");

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    for (int i = 0; i < deviceCnt; i++) {
        hipFree(a_d[i]);
        hipFree(b_d[i]);
        hipFree(c_d[i]);
    }
}

void thread_gpu (int size) {
    int n = size;
    int nBytes = n * sizeof(float);

    float *a, *b, *c;
    float *a_d, *b_d, *c_d;

    int deviceCnt;
    hipGetDeviceCount(&deviceCnt);
    printf("Devices: %d\n", deviceCnt);

    dim3 block(BLOCKSIZE);
    dim3 grid((unsigned int)ceil(n/(float)block.x) / deviceCnt);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&start);

    hipHostAlloc(&a, nBytes, 0);
    hipHostAlloc(&b, nBytes, 0);
    hipHostAlloc(&c, nBytes, 0);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    int n_per_device = n / deviceCnt + 1;
    int bytes_per_device = nBytes / deviceCnt + 1;
#pragma omp parallel num_threads(deviceCnt)
    {
        int device = omp_get_thread_num();
        hipSetDevice(device);
        hipMalloc(&a_d, bytes_per_device);
        hipMalloc(&b_d, bytes_per_device);
        hipMalloc(&c_d, bytes_per_device);

        hipEventRecord(start);
        hipMemcpy(a_d, a + device * n_per_device, bytes_per_device, hipMemcpyHostToDevice);
        hipMemcpy(b_d, c + device * n_per_device, bytes_per_device, hipMemcpyHostToDevice);

        vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n_per_device);
        hipMemcpy(c + device * n_per_device, c_d, bytes_per_device, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        printf("Elapsed time: %f\n", ms);
        hipFree(a_d);
        hipFree(b_d);
        hipFree(c_d);
    }

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}

int main(int argc, char **argv) {
    assert(argc==3);
    switch (atoi(argv[2])) {
    case 0:
        usual_sample(atoi(argv[1]));
        break;
    case 1:
        pinned_sample(atoi(argv[1]));
        break;
    case 2:
        unified_sample(atoi(argv[1]));
        break;
    case 3:
        multigpu(atoi(argv[1]));
        break;
    default:
        break;
    }
    return 0;
}
