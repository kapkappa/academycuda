#include "hip/hip_runtime.h"
// High level matrix multiplication on GPU using CUDA with Thrust, CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <sys/time.h>
double timer() {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const thrust::device_vector<float> &A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main(int argc, char**argv) {
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
    assert(argc==2);
    int n = atoi(argv[1]);
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = n;

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A), d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);

	// Optionally we can print the data
//	std::cout << "A =" << std::endl;
//	print_matrix(d_A, nr_rows_A, nr_cols_A);
//	std::cout << "B =" << std::endl;
//	print_matrix(d_B, nr_rows_B, nr_cols_B);

	// Multiply A and B on GPU
    double t1 = timer();
	gpu_blas_mmul(thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]), thrust::raw_pointer_cast(&d_C[0]), nr_rows_A, nr_cols_A, nr_cols_B);
    double t2 = timer();
    std::cout << "Sgemm time: " << t2-t1 << std::endl;

    double it = n / 1024;
    double oper_num = 2 * it * it * it;
    std::cout << "Efficiency: " << oper_num / (t2-t1) << " GFLOPS" << std::endl;

	//Print the result
//	std::cout << "C =" << std::endl;
//	print_matrix(d_C, nr_rows_C, nr_cols_C);

	return 0;
}
