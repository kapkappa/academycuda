#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void vectorAddGPU(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_sample (int size = 1048576) {
    int n = size;
    int nBytes = n * sizeof(float);

    float *a, *b, *c;

    hipEvent_t uniStart, uniStop;
    hipEventCreate(&uniStart);
    hipEventCreate(&uniStop);

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

//    printf("UNI: allocating memory\n");
    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);
    hipMallocManaged(&c, nBytes);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEventRecord(uniStart);

    vectorAddGPU<<<grid, block>>>(a, b, c, n);

    hipEventRecord(uniStop);
    hipDeviceSynchronize();

    float ms = 0;
    hipEventElapsedTime(&ms, uniStart, uniStop);
    printf("UNI: Memalloc(unified memory) + Kernel time is: %f\n", ms);
    hipDeviceSynchronize();
}

void pinned_sample (int size = 1048576) {
    int n = size;
    size_t nBytes = n * sizeof(float);
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
//    float errNorm, refNorm, ref, diff;

    hipEvent_t pinStart, pinStop;
    hipEventCreate(&pinStart);
    hipEventCreate(&pinStop);

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);
    hipMalloc(&d_a, nBytes);
    hipMalloc(&d_b, nBytes);
    hipMalloc(&d_c, nBytes);

    for (int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEventRecord(pinStart);

    hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, nBytes, hipMemcpyHostToDevice);
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, n);

    hipEventRecord(pinStop);
    hipDeviceSynchronize();

    float ms = 0;
    hipEventElapsedTime(&ms, pinStart, pinStop);
    printf("PIN: Memcpy + Kernel time is: %f\n", ms);
    hipDeviceSynchronize();
}

void usual_sample (int size = 1048576) {
    int n = size;
    int nBytes = n*sizeof(float);

    float *a, *b;  // host data
    float *c;  // results

    a = (float*)malloc(nBytes);
    b = (float*)malloc(nBytes);
    c = (float*)malloc(nBytes);

    float *a_d,*b_d,*c_d;

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i = 0; i < n; i++) {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("Allocating device memory on host..\n");

    hipEvent_t start, stop, malloc_start, malloc_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&malloc_start);
    hipEventCreate(&malloc_stop);

    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));

    printf("Copying to device..\n");

    hipEventRecord(start);

    hipMemcpy(a_d, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(float), hipMemcpyHostToDevice);

    printf("Doing GPU Vector add\n");

    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

    hipEventRecord(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("USUAL Memcpy + Kernel: %f ms\n", milliseconds);

    hipDeviceSynchronize();

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}


int main(int argc, char **argv) {
    assert(argc==2);
    usual_sample(atoi(argv[1]));
    pinned_sample(atoi(argv[1]));
    unified_sample(atoi(argv[1]));

    return 0;
}
