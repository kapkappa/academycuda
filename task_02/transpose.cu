#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <assert.h>
#include <sys/time.h>

const int B_SIZE = 1024;

double timer() {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

__global__ void transpose(double *M, double *M_t, int n) {

    int Idx = blockIdx.x * B_SIZE + threadIdx.x;
//    int y = blockIdx.y * B_SIZE + threadIdx.y;
//    int Width = gridDim.x * B_SIZE;

//    int global = y * Width + x;
//    if (global < n * n) return;

//    for (int j = 0; j < blockDim.x; j += blockDim.y) {
//        M_t[x * Width + (y+j)] = M[(y+j) * Width + x];
//    }

    if(Idx < n*n) {
        int x = Idx % n;
        int y = Idx / n;
        int T_index = x * n + y;
        M_t[T_index] = M[Idx];
    }
}

void print(double *M, int n) {
    std::cout << "Print matrix: \n";
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            std::cout << M[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main(int argc, char ** argv) {
    assert(argc == 2);
    int n = atoi(argv[1]);
    size_t size = n * n * sizeof(double);
    double *M = (double*)malloc(size);

    //init
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            M[i * n + j] = i / j + j % i;
        }
    }

    double *M_t = (double*)malloc(size);

    double t1 = timer();
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            M_t[j * n + i] = M[i * n + j];
        }
    }

    double t2 = timer();
    std::cout << "CPU time: " << t2-t1 << std::endl;

    print(M, n);
    print(M_t, n);

    dim3 block(B_SIZE);
    dim3 grid((n * n - 1) / B_SIZE + 1);

    double *M_dev, *M_t_dev;
    hipMalloc(&M_dev, size);
    hipMalloc(&M_t_dev, size);

    hipMemcpy(M_dev, M, size, hipMemcpyHostToDevice);

    transpose<<<grid, block>>>(M_dev, M_t_dev, n);

    hipMemcpy(M_t, M_t_dev, size, hipMemcpyDeviceToHost);

    print(M_t, n);

    free(M);
    free(M_t);

    hipFree(M_dev);
    hipFree(M_t_dev);

    return 0;
}
